#include <stdlib.h>
#include <stdio.h>

#include <iostream>

#include <hip/hip_runtime.h>


void _CheckCudaError(const hipError_t hipError_t, const char* file, const int line)
{
	if (hipError_t != hipSuccess) {
		std::cout << "[CUDA ERROR] " << hipGetErrorString(hipError_t) << " (" << file << ":" << line << ")\n";
		exit(EXIT_FAILURE);
	}
}
#define CheckCudaError(call) _CheckCudaError((call), __FILE__, __LINE__)

__global__ void add(const int *a, const int *b, int *c)
{
	int tid = blockIdx.x;

	if (tid < gridDim.x)
		c[tid] = a[tid] + b[(gridDim.x - 1)- tid];
}

int main()
{
	int h_a[6] = { 1, 2, 3, 4, 5, 6 };
	int h_b[6] = { 10, 20, 30, 40, 50, 60 };
	int h_c[6];

	int* d_a;
	int* d_b;
	int* d_c;

	CheckCudaError(hipMalloc(&d_a, 6 * sizeof(int)));
	CheckCudaError(hipMalloc(&d_b, 6 * sizeof(int)));
	CheckCudaError(hipMalloc(&d_c, 6 * sizeof(int)));

	CheckCudaError(hipMemcpy(d_a, h_a, 6 * sizeof(int), hipMemcpyHostToDevice));
	CheckCudaError(hipMemcpy(d_b, h_b, 6 * sizeof(int), hipMemcpyHostToDevice));

	add<<<6, 1>>>(d_a, d_b, d_c);
	CheckCudaError(hipDeviceSynchronize());

	CheckCudaError(hipMemcpy(h_c, d_c, 6 * sizeof(int), hipMemcpyDeviceToHost));
	for (int i = 0; i < 6; i++)
		printf("%d ", h_c[i]);
	printf("\n");

	CheckCudaError(hipFree(d_a));
	CheckCudaError(hipFree(d_b));
	CheckCudaError(hipFree(d_c));
}