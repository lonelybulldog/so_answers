#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>

void _CheckCudaError(hipError_t ret, char *file, int line)
{
	if (ret != hipSuccess) {
		printf("%s - %s (%s:%d)\n", hipGetErrorName(ret), hipGetErrorString(ret), file, line);
		exit(EXIT_FAILURE);
	}
}
#define CheckCudaError(call)	_CheckCudaError((call), __FILE__, __LINE__)

template <typename T>
__global__ void copyKernel(const T* __restrict__ a, T* __restrict__ b)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
	b[i] = a[i];
}

template <typename T>
void test()
{
	// test up to 1M element
	const int arraySize = 1024 * 1024;
	const int memSize = arraySize * sizeof(T);

	T* a = (T*)malloc(memSize);
	T* b = (T*)malloc(memSize);

	for (int i = 0; i < arraySize; i++)
		a[i] = i % 256;                                 // value cap at max value of unsigned char type

	T* d_a;
	T* d_b;

	CheckCudaError(hipMalloc(&d_a, memSize));
	CheckCudaError(hipMalloc(&d_b, memSize));

	CheckCudaError(hipMemcpy(d_a, a, memSize, hipMemcpyHostToDevice));

	// test from 1 threads to 31 threads
	for (int i = 1; i <= 31; i++)
		copyKernel<T><<<1, i>>>(d_a, d_b);

	// test from 32 threads to 1K threads
	for (int i = 32; i <= 1024; i *= 2) {
		CheckCudaError(hipMemset(d_b, 0, memSize));    // reset device memory of b to 0

		copyKernel<T><<<1, i>>>(d_a, d_b);

		CheckCudaError(hipMemcpy(b, d_b, memSize, hipMemcpyDeviceToHost));

		for (int j = 0; j < i; j++)
			if (b[j] != j%256) {                        // if something is wrong with kernel
				printf("i = %d, j = %d, b[j] = %d\n", i, j, b[j]);
				exit(EXIT_FAILURE);
			} else
				b[j] = 0;                               // reset host memory of b to 0
	} 

	// test from 1K threads to 1M threads
	for (int i = 1024; i <= 1024 * 1024; i *= 2) {
		CheckCudaError(hipMemset(d_b, 0, memSize));    // reset device memory of b to 0

		copyKernel<T><<<1024, i/1024>>>(d_a, d_b);

		CheckCudaError(hipMemcpy(b, d_b, memSize, hipMemcpyDeviceToHost));

		for (int j = 0; j < i; j++)
			if (b[j] != j%256) {                        // if something is wrong with kernel
				printf("i = %d, j = %d, b[j] = %d\n", i, j, b[j]);
				exit(EXIT_FAILURE);
			} else
				b[j] = 0;                               // reset host memory of b to 0
	} 

}

int main()
{
	test<unsigned char>();
	test<int>();

	CheckCudaError(hipDeviceReset());
}