#include "hip/hip_runtime.h"
/*
 * Compile      : nvcc main.cu -lX11
 *
 * Prerequisite : sudo apt install cimg-dev
 */
#include "hip/hip_runtime.h"
#include ""

#include "CImg.h"
#include <iostream>

using namespace std;
using namespace cimg_library;

__global__ void rgb2gray(unsigned char * d_src, unsigned char * d_dst, int width, int height)
{
	int pos_x = blockIdx.x * blockDim.x + threadIdx.x;
	int pos_y = blockIdx.y * blockDim.y + threadIdx.y;

	if (pos_x >= width || pos_y >= height)
	    return;

	unsigned char r = d_src[pos_y * width + pos_x];
	unsigned char g = d_src[(height + pos_y ) * width + pos_x];
	unsigned char b = d_src[(height * 2 + pos_y) * width + pos_x];

	unsigned int _gray = (unsigned int)((float)(r + g + b) / 3.0f + 0.5);
	unsigned char gray = _gray > 255 ? 255 : _gray;

	d_dst[pos_y * width + pos_x] = gray;
}


int main()
{
    //Load image
    CImg<unsigned char> src("lena.gif");
    int width = src.width();
    int height = src.height();
    unsigned long sizee = src.size();

    int sze = width * height;

    cout << sze << endl;

    //create pointer to image
    unsigned char *h_src = src.data();

    CImg<unsigned char> dst(width, height, 1, 1);
    unsigned char *h_dst = dst.data();

    unsigned char *d_src;
    unsigned char *d_dst;

    cout << sizee << endl;

    hipMalloc((void**)&d_src, sizee);
    hipMalloc((void**)&d_dst, width*height*sizeof(unsigned char));

    hipMemcpy(d_src, h_src, sizee, hipMemcpyHostToDevice);

    //launch the kernel
	dim3 blkDim (16, 16, 1);
	dim3 grdDim ((width + 15)/16, (height + 15)/16, 1);
	rgb2gray<<<grdDim, blkDim>>>(d_src, d_dst, width, height);

    //force the printf()s to flush
    hipDeviceSynchronize();

    // copy back the result array to the CPU
    hipMemcpy(h_dst, d_dst, width*height, hipMemcpyDeviceToHost);

    hipFree(d_src);
    hipFree(d_dst);

    CImgDisplay main_disp(dst, "After Processing");
    while (!main_disp.is_closed())
        main_disp.wait();

    return 0;
}
