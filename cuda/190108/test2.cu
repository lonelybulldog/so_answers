
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void kernel2(int *data)
{
	__shared__ volatile int data_s[32];

	size_t t_id = threadIdx.x;

	data_s[t_id] = data[t_id];

	if (1 <= t_id)
		data_s[t_id] += data_s[t_id - 1];
	if (2 <= t_id)
		data_s[t_id] += data_s[t_id - 2];
	if (4 <= t_id)
		data_s[t_id] += data_s[t_id - 4];
	if (8 <= t_id)
		data_s[t_id] += data_s[t_id - 8];
	if (16 <= t_id)
		data_s[t_id] += data_s[t_id - 16];

	data[t_id] = data_s[t_id];
}

int main()
{
	int data[32];
	int result[32];

	int *data_d;
	hipMalloc(&data_d, sizeof(data));

	for (int i = 0; i < 32; i++)
		data[i] = i;

	dim3 gridDim(1);
	dim3 blockDim(32);

	hipMemcpy(data_d, data, sizeof(data), hipMemcpyHostToDevice);
	kernel2<<<gridDim, blockDim>>>(data_d);
	hipMemcpy(result, data_d, sizeof(data), hipMemcpyDeviceToHost);

	printf("kernel2 : ");
	for (int i = 0; i < 32; i++)
		printf("%4i ", result[i]);
	printf("(device, shared memory with volatile qualifier)\n");
}
