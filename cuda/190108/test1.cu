
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void kernel1(int *data)
{
	__shared__ int data_s[32];

	size_t t_id = threadIdx.x;

	data_s[t_id] = data[t_id];

	if (1 <= t_id)
		data_s[t_id] += data_s[t_id - 1];
	if (2 <= t_id)
		data_s[t_id] += data_s[t_id - 2];
	if (4 <= t_id)
		data_s[t_id] += data_s[t_id - 4];
	if (8 <= t_id)
		data_s[t_id] += data_s[t_id - 8];
	if (16 <= t_id)
		data_s[t_id] += data_s[t_id - 16];

	data[t_id] = data_s[t_id];
}

int main()
{
	int data[32];
	int result[32];

	int *data_d;
	hipMalloc(&data_d, sizeof(data));

	for (int i = 0; i < 32; i++)
		data[i] = i;

	dim3 gridDim(1);
	dim3 blockDim(32);

	hipMemcpy(data_d, data, sizeof(data), hipMemcpyHostToDevice);
	kernel1<<<gridDim, blockDim>>>(data_d);
	hipMemcpy(result, data_d, sizeof(data), hipMemcpyDeviceToHost);

	printf("kernel1 : ");
	for (int i = 0; i < 32; i++)
		printf("%4i ", result[i]);
	printf("\n");
}
