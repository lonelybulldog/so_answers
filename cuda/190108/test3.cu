
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void kernel3(int *data)
{
	__shared__ int data_s[32];

	size_t t_id = threadIdx.x;

	data_s[t_id] = data[t_id];

	int v = data_s[t_id];           __syncwarp();

	if (1 <= t_id) {
		v += data_s[t_id - 1];  __syncwarp();
		data_s[t_id] = v;       __syncwarp();
	}
	if (2 <= t_id) {
		v += data_s[t_id - 2];  __syncwarp();
		data_s[t_id] = v;       __syncwarp();
	}
	if (4 <= t_id) {
		v += data_s[t_id - 4];  __syncwarp();
		data_s[t_id] = v;       __syncwarp();
	}
	if (8 <= t_id) {
		v += data_s[t_id - 8];  __syncwarp();
		data_s[t_id] = v;       __syncwarp();
	}
	if (16 <= t_id) {
		v += data_s[t_id - 16]; __syncwarp();
		data_s[t_id] = v;
	}

	data[t_id] = data_s[t_id];
}

int main()
{
	int version;
	hipRuntimeGetVersion(&version);
	if (version < 9000) {
		printf("Test not supported in this CUDA runtime version (%d)\n", version);
		exit(EXIT_SUCCESS);
	}

	int data[32];
	int result[32];

	int *data_d;
	hipMalloc(&data_d, sizeof(data));

	for (int i = 0; i < 32; i++)
		data[i] = i;

	dim3 gridDim(1);
	dim3 blockDim(32);

	hipMemcpy(data_d, data, sizeof(data), hipMemcpyHostToDevice);
	kernel3<<<gridDim, blockDim>>>(data_d);
	hipMemcpy(result, data_d, sizeof(data), hipMemcpyDeviceToHost);

	printf("kernel3 : ");
	for (int i = 0; i < 32; i++)
		printf("%4i ", result[i]);
	printf("(device, shared memory with __syncwarp(FULL_MASK))\n");
}
