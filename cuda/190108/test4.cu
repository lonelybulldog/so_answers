
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void kernel4(int *data)
{
	__shared__ int data_s[32];

	size_t t_id = threadIdx.x;

	data_s[t_id] = data[t_id];

	int v = data_s[t_id];

	unsigned mask = 0xffffffff;     __syncwarp(mask);

	mask = __ballot_sync(0xffffffff, 1 <= t_id);
	if (1 <= t_id) {
		v += data_s[t_id - 1];  __syncwarp(mask);
		data_s[t_id] = v;       __syncwarp(mask);
	}
	mask = __ballot_sync(0xffffffff, 2 <= t_id);
	if (2 <= t_id) {
		v += data_s[t_id - 2];  __syncwarp(mask);
		data_s[t_id] = v;       __syncwarp(mask);
	}
	mask = __ballot_sync(0xffffffff, 4 <= t_id);
	if (4 <= t_id) {
		v += data_s[t_id - 4];  __syncwarp(mask);
		data_s[t_id] = v;       __syncwarp(mask);
	}
	mask = __ballot_sync(0xffffffff, 8 <= t_id);
	if (8 <= t_id) {
		v += data_s[t_id - 8];  __syncwarp(mask);
		data_s[t_id] = v;       __syncwarp(mask);
	}
	mask = __ballot_sync(0xffffffff, 16 <= t_id);
	if (16 <= t_id) {
		v += data_s[t_id - 16]; __syncwarp(mask);
		data_s[t_id] = v;
	}

	data[t_id] = data_s[t_id];
}

int main()
{
	int version;
	hipRuntimeGetVersion(&version);
	if (version < 9000) {
		printf("Test not supported in this CUDA runtime version (%d)\n", version);
		exit(EXIT_SUCCESS);
	}

	int data[32];
	int result[32];

	int *data_d;
	hipMalloc(&data_d, sizeof(data));

	for (int i = 0; i < 32; i++)
		data[i] = i;

	dim3 gridDim(1);
	dim3 blockDim(32);

	hipMemcpy(data_d, data, sizeof(data), hipMemcpyHostToDevice);
	kernel4<<<gridDim, blockDim>>>(data_d);
	hipMemcpy(result, data_d, sizeof(data), hipMemcpyDeviceToHost);

	printf("kernel4 : ");
	for (int i = 0; i < 32; i++)
		printf("%4i ", result[i]);
	printf("(device, shared memory with masked __syncwarp())\n");
}
